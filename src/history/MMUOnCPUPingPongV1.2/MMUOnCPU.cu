#include "hip/hip_runtime.h"
#include "MMUOnCPU.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <pthread.h>



/* -- Implementation of memory management with linked lists -- */
LinkedListManagement::LinkedListManagement(void * startAddressIn)
{
    //init start address
    startAddress = startAddressIn;

    //create a init blanket Node
    MemorySegmentNode* newMemorySegmentNode = (MemorySegmentNode *)malloc(sizeof(MemorySegmentNode));
    if (newMemorySegmentNode == NULL) {
        fprintf(stderr, "Error allocating memory for list node\n");
        exit(EXIT_FAILURE);
    }
    newMemorySegmentNode->length = MAX_LINKEDLIST_MEM_SIZE;
    newMemorySegmentNode->offsetAddress = 0;
    newMemorySegmentNode->next = NULL;
    newMemorySegmentNode->isHole = true;
    
    ListHead = newMemorySegmentNode;

}

LinkedListManagement::~LinkedListManagement()
{
    //free the entire list of MemorySegmentNode
    MemorySegmentNode* currentNode = ListHead;
    MemorySegmentNode* nextNode;
    while(currentNode != NULL){
        nextNode = currentNode -> next;
        free(currentNode);
        currentNode = nextNode;
    }
    ListHead = NULL;

}

MemorySegmentNode* LinkedListManagement::findSuitableMem(unsigned int sizeAllocate)
{
    //return the current node when finding a suitable node
    MemorySegmentNode* returnNode = NULL;
    MemorySegmentNode* currentNode = ListHead;
    while (currentNode != NULL) {
        if((currentNode -> isHole == true) && (currentNode ->length >= sizeAllocate))
        {
            return currentNode;
        }
        currentNode = currentNode->next;
    }
    return returnNode;
}

void* LinkedListManagement::allocateMem(unsigned int sizeAllocate)
{
    void* returnAddress = NULL;
    MemorySegmentNode* insertNode = NULL;
    //find a suitable position to allocate and return the pointer to the last node.
    insertNode = findSuitableMem(sizeAllocate);
    
    //if cannot find a suitable memory block
    if (insertNode == NULL){
        fprintf(stderr, "Error allocating memory for memory\n");
        exit(EXIT_FAILURE);
    }

    /*------Update memory management nodes--------*/
    returnAddress = (void*)((char*)startAddress + insertNode -> offsetAddress);
    //If the allocated length is equal to memory block existed, just chage the status of memory management node
    //isHole to false.
    if(insertNode->length == sizeAllocate)
    {
        insertNode -> isHole = false;
        return returnAddress;
    }

    //create a new Node
    MemorySegmentNode* newMemorySegmentNode = (MemorySegmentNode *)malloc(sizeof(MemorySegmentNode));
    if (newMemorySegmentNode == NULL) {
        fprintf(stderr, "Error allocating memory for list node\n");
        exit(EXIT_FAILURE);
    }

    newMemorySegmentNode -> length = insertNode -> length - sizeAllocate;
    newMemorySegmentNode -> offsetAddress = insertNode -> offsetAddress + sizeAllocate;
    newMemorySegmentNode -> next = insertNode -> next;
    newMemorySegmentNode -> isHole = true;

    //upadate original nodes
    insertNode -> isHole = false;
    insertNode -> length = sizeAllocate;
    insertNode -> next = newMemorySegmentNode;
    
    return returnAddress;
}

void LinkedListManagement::freeMem(void* addressIn)
{
    /*--the lack of check the avaliable scope of addressIn.-- */
    unsigned int currentAddressOffset = (unsigned int)((char*)addressIn - (char*)startAddress);
    //Find the corresponding node
    MemorySegmentNode* currentNode = ListHead;
    MemorySegmentNode* lastNode = NULL;
    MemorySegmentNode* nextNode = NULL;
    while (currentNode != NULL) {
        if (currentNode->offsetAddress == currentAddressOffset) {
            break;
        }
        lastNode = currentNode;
        currentNode = currentNode->next;
    }

    nextNode = currentNode->next;
    //check the results
    // if (currentNode == NULL){
    //     /*---if cannot find it, return an error----*/
    //     return;
    // }

    /* -- merge the neighbor idle memory blocks and clean the unnecessary management nodes*/
    if(lastNode == NULL)
    {
        if(nextNode == NULL)
        {
            currentNode -> isHole = true;
        }else
        {
            if(nextNode ->isHole == false){
                currentNode -> isHole = true;
            }else{
                currentNode -> isHole = true;
                currentNode -> length = nextNode -> length + currentNode -> length;
                currentNode -> next = nextNode -> next;
                free(nextNode);
            }
        }
        return;
    }

    if(nextNode == NULL)
    {
        if(lastNode -> isHole == true)
        {
            lastNode -> next = nextNode;
            lastNode -> length = lastNode -> length + currentNode -> length;
            free(currentNode);
        }else{
            currentNode -> isHole = true;
        }
        return;
    }

    if(lastNode-> isHole == true)
    {
        lastNode -> next = nextNode;
        lastNode -> length = lastNode -> length + currentNode -> length;
        free(currentNode);
        if(nextNode -> isHole == true)
        {
            lastNode -> next = nextNode -> next;
            lastNode -> length = lastNode -> length + nextNode -> length;
            free(nextNode);
        }
        //else if nextNode's isHole is false, do nothing
    }else
    {
        if(nextNode -> isHole == true)
        {
            currentNode -> isHole = true;
            currentNode -> length = nextNode -> length + currentNode -> length;
            currentNode -> next = nextNode -> next;
            free(nextNode);
        }else{
            currentNode -> isHole = true;
        }
    }

}
/* -- End -- Implementation of memory management with linked lists -- */


/* -- Implementation of memory management with bitmaps -- */
BitmapManagement::BitmapManagement(void * startAddressIn)
{
    //init start address
    startAddress = startAddressIn;

    //init bitmap
    initializeBitmap();

    //init hash tables
    initializeHashTable();
}

BitmapManagement::~BitmapManagement()
{
    freeHashTable();
}

void BitmapManagement::initializeBitmap()
{
    memset(bitmap, 0, sizeof(bitmap));
    return;
}

void BitmapManagement::setBitMapTrue(unsigned int blockIndex)
{
    unsigned int byteIndex = blockIndex / 8;
    unsigned int bitIndex = blockIndex % 8;
    unsigned char mask = 1 << bitIndex;
    bitmap[byteIndex] |= mask;
    return;
}

void BitmapManagement::setBitsMapTrue(unsigned int size, unsigned int startBlockIndex)
{
    unsigned int endBlockIndex = startBlockIndex + size;
    unsigned int startByteIndex = startBlockIndex / 8;
    unsigned int endByteIndex = (endBlockIndex - 1) / 8;

    if (startByteIndex == endByteIndex)
    {
        // If size is no larger than 8 and within a single byte
        unsigned int startBitIndex = startBlockIndex % 8;
        unsigned int endBitIndex = endBlockIndex % 8;
        unsigned char mask = ((0xFF >> (8 - endBitIndex)) & (0xFF << startBitIndex));

        bitmap[startByteIndex] |= mask;
        return;
    }

    for (unsigned int i = startByteIndex; i <= endByteIndex; i++) {
        unsigned char mask;

        if (i == startByteIndex) {
            // Create mask for the first byte
            unsigned int startBitIndex = startBlockIndex % 8;
            mask = (0xFF << startBitIndex);
        } else if (i == endByteIndex) {
            // Create mask for the last byte
            unsigned int endBitIndex = endBlockIndex % 8;
            if(endBitIndex == 0)
                endBitIndex = endBitIndex + 8;
            mask = (0xFF >> (8 - endBitIndex));
        } else {
            // Set all bits in the intermediate bytes
            mask = 0xFF;
        }

        bitmap[i] |= mask;
    }
    return;
}

void BitmapManagement::clearBit(unsigned int blockIndex)
{
    unsigned int byteIndex = blockIndex / 8;
    unsigned int bitIndex = blockIndex % 8;
    unsigned char mask = ~(1 << bitIndex);
    bitmap[byteIndex] &= mask;
}

void BitmapManagement::clearBits(unsigned int size, unsigned int startBlockIndex)
{
    unsigned int endBlockIndex = startBlockIndex + size;
    unsigned int startByteIndex = startBlockIndex / 8;
    unsigned int endByteIndex = (endBlockIndex - 1) / 8;

    if (startByteIndex == endByteIndex)
    {
        // If size is no larger than 8 and within a single byte
        unsigned int startBitIndex = startBlockIndex % 8;
        unsigned int endBitIndex = endBlockIndex % 8;
        unsigned char mask = ~((0xFF >> (8 - endBitIndex)) & (0xFF << startBitIndex));

        bitmap[startByteIndex] &= mask;
        return;
    }

    // Handle multiple bytes
    for (unsigned int i = startByteIndex; i <= endByteIndex; i++) 
    {
        unsigned char mask;
        if (i == startByteIndex) {
            // Create mask for the first byte
            unsigned int startBitIndex = startBlockIndex % 8;
            mask = ~(0xFF << startBitIndex);
        } else if (i == endByteIndex) {
            // Create mask for the last byte
            unsigned int endBitIndex = endBlockIndex % 8;
            if(endBitIndex == 0)
                endBitIndex = endBitIndex + 8;
            mask = ~(0xFF >> (8 - endBitIndex));
        } else {
            // Clear all bits in the intermediate bytes
            mask = 0x00;
        }

        bitmap[i] &= mask;
    }
    return;
}

bool BitmapManagement::isBitSet(unsigned int blockIndex)
{
    unsigned int byteIndex = blockIndex / 8;
    unsigned int bitIndex = blockIndex % 8;
    unsigned char mask = 1 << bitIndex;
    return (bool)(bitmap[byteIndex] & mask) != 0;    
}

bool BitmapManagement::isZeroBit(unsigned char byte, unsigned int position) 
{
    return (byte & (1 << position)) == 0;
}

unsigned int BitmapManagement::findPage()
{
    //Simple sequential search
    for(unsigned int i =0; i < BITMAP_SIZE; i++)
    {
        if (!isBitSet(i))
        {
            return i; // Return the index of the allocated block
        }
    }
    return 0;
}

unsigned int BitmapManagement::findPages(unsigned int pageSize)
{
    //find continuous pages
    //Simple sequential search
    unsigned int count = 0; // Count of continuous zeros
    for (unsigned int i = 0; i < PAGE_NUMBER; ++i) {
        if (isZeroBit(bitmap[i / 8], i % 8)) {
            count++;
            if (count == pageSize) {
                return i - pageSize + 1; // Return the starting index of the sequence
            }
        } else {
            count = 0; // Reset count if a one is found
        }
    }
    return 0; // Return 0 if not found
}

void* BitmapManagement::allocateMem(unsigned int sizeAllocate)
{
    //get the size of continuous pages
    unsigned int pages = ceil(float(sizeAllocate) / PAGE_SIZE);
    
    //if just need only one page, just find a idle block
    if (pages == 1)
    {  
        unsigned int targetBlockIndex = findPage();
        setBitMapTrue(targetBlockIndex);
        insertHashTable(targetBlockIndex, pages);
        return (void*)((char*)startAddress + targetBlockIndex*PAGE_SIZE);
    }

    //more than one pages
    unsigned int startBlockIndex = findPages(pages);
    setBitsMapTrue(pages, startBlockIndex);

    //update block-size table
    insertHashTable(startBlockIndex, pages);

    return (void*)((char*)startAddress + startBlockIndex*PAGE_SIZE);

}

void BitmapManagement::freeMem(void* addressIn)
{
    //get the size of blockID
    unsigned int offsetAddress = (unsigned int)((char*)addressIn - (char*)startAddress);
    unsigned int blockID = offsetAddress / PAGE_SIZE;

    //search the blockID and its size
    unsigned pageNumber = findHashTable(blockID);
    //if just free only one page, just clear the block
    if(pageNumber == 1)
    {
        clearBit(pageNumber);  
    }else
    {
        clearBits(pageNumber, blockID);
    }
    
    deleteHashTable(blockID);
    return;
      
}

unsigned int BitmapManagement::hash(unsigned int key)
{
    key = ((key >> 16) ^ key) * 0x45d9f3b;
    key = ((key >> 16) ^ key) * 0x45d9f3b;
    key = (key >> 16) ^ key;
    return key & (HASHTABLE_BITMAP_BLOCKIDSIZE_SIZE - 1); // TABLE_SIZE is 4096
}

void BitmapManagement::initializeHashTable()
{
    for (int i = 0; i < HASHTABLE_BITMAP_BLOCKIDSIZE_SIZE; ++i) {
        buckets[i] = NULL;
    }
}

void BitmapManagement::insertHashTable(unsigned int key, unsigned int value)
{
    unsigned int idx = hash(key);
    BlockIDSizeNode* newNode = (BlockIDSizeNode*)malloc(sizeof(BlockIDSizeNode));

    newNode->pair.blockID = key;
    newNode->pair.size = value;

    // Insert at the beginning of the chain (bucket)
    newNode->next = buckets[idx];
    buckets[idx] = newNode;
}

unsigned int BitmapManagement::findHashTable(unsigned int key)
{
    unsigned int idx = hash(key);
    unsigned int returnValue;
    BlockIDSizeNode* current = buckets[idx];
    while (current != NULL) {
        if (current->pair.blockID == key)
        {
            returnValue = current->pair.size;
            return returnValue;
        }
        current = current->next;
    }
    return 0;
}

void BitmapManagement::deleteHashTable(unsigned int key)
{
    unsigned int idx = hash(key);
    BlockIDSizeNode *current = buckets[idx];
    BlockIDSizeNode *prev = NULL;
    while (current != NULL) {
        if (current->pair.blockID == key) {
            if (prev == NULL) {
                buckets[idx] = current->next;
            } else {
                prev->next = current->next;
            }
            free(current);
            return;
        }
        prev = current;
        current = current->next;
    }
}

void BitmapManagement::freeHashTable()
{
    for (int i = 0; i < HASHTABLE_BITMAP_BLOCKIDSIZE_SIZE; ++i) {
        BlockIDSizeNode* current = buckets[i];
        while (current != NULL) {
            BlockIDSizeNode* temp = current;
            current = current->next;
            free(temp);
        }
    }
}
/* -- End -- Implementation of memory management with bitmaps -- */


/* -- Implementation of memory management -- */
MemoryManagement::MemoryManagement()
{
    //init startAddress

    // bitmapStartAddress = (void*)malloc(MAX_BITMAP_MEM_SIZE);
    // linkedListStartAddress = (void*)malloc(MAX_LINKEDLIST_MEM_SIZE);

    size_t size = 1024*1024*1024;
    hipMallocManaged(&bitmapStartAddress, size);
    size = 1024*1024*1024;
    hipMallocManaged(&linkedListStartAddress, size);

    //init class BitmapManagement and LinkedListManagement
    bitmapManagement = new BitmapManagement(bitmapStartAddress);
    linkedListManagement = new LinkedListManagement(linkedListStartAddress);
}

MemoryManagement::~MemoryManagement()
{
    delete bitmapManagement;
    delete linkedListManagement;

    // free(bitmapStartAddress);
    // free(linkedListStartAddress);

    hipFree(bitmapStartAddress);
    hipFree(linkedListStartAddress);
}

void MemoryManagement::allocateMem(struct MMUOnTransfer* pMMUOnTransfer, int bid)
{
    int startBlock = bid * BLOCKSIZE;
    for(int i = 0; i < BLOCKSIZE; i++)
    {
        int tmpi = startBlock + i;
        if(pMMUOnTransfer -> sizeAllocate[tmpi]  * 8 > THRESHOLD_MEMSIZE)
        {
            pMMUOnTransfer -> addressAllocate[tmpi] = linkedListManagement->allocateMem(pMMUOnTransfer -> sizeAllocate[tmpi]);
        }else
        {
            pMMUOnTransfer -> addressAllocate[tmpi] = bitmapManagement->allocateMem(pMMUOnTransfer -> sizeAllocate[tmpi]);
        }
    }
}

void MemoryManagement::freeMem(struct MMUOnTransfer* pMMUOnTransfer, int bid)
{
    void* bitmapEndAddress = (void*)((char*)(bitmapStartAddress) + MAX_BITMAP_MEM_SIZE);
    int startBlock = bid * BLOCKSIZE;
    for(int i = 0; i < BLOCKSIZE; i++)
    {
        int tmpi = startBlock + i;
        if((pMMUOnTransfer -> addressFree[tmpi]) >= bitmapStartAddress && (pMMUOnTransfer -> addressFree[tmpi]) < bitmapEndAddress)
        {
            bitmapManagement->freeMem(pMMUOnTransfer -> addressFree[tmpi]);
        }else
        {
            linkedListManagement->freeMem(pMMUOnTransfer -> addressFree[tmpi]);
        }
    }
}

void* MemoryManagement::getBitmapStartAddress()
{
    return bitmapStartAddress;
}

void* MemoryManagement::getLinkedListStartAddress()
{
    return linkedListStartAddress;
}
/* -- End -- Implementation of memory management -- */

/* -- allocation function*/
//launch threads to deal with blocks allocation and free
void* threadAllocation(void* arg)
{
    thread_args* args = (thread_args*)arg;
    
    //create threads for block allocations
    if(BLOCKNUMBER < ALLOCATIONMANAGEMENTTHREADNUMBER)
    {
        // if threads smaller than blocknumber
        // Create and start threads
        pthread_t threads[BLOCKNUMBER];
        threadBlockAllocations pthreadBlockAllocations[BLOCKNUMBER];
        for(int i = 0; i < BLOCKNUMBER; ++i) 
        {
            pthreadBlockAllocations[i].targs = args;
            pthreadBlockAllocations[i].start = i;
            pthreadBlockAllocations[i].end = i + 1;

            pthread_create(&threads[i], NULL, blockAllocationThr, &pthreadBlockAllocations[i]);
        }

        for (int i = 0; i < BLOCKNUMBER; ++i) {
            pthread_join(threads[i], NULL);
        }
    }else{
        pthread_t threads[ALLOCATIONMANAGEMENTTHREADNUMBER];
        threadBlockAllocations pthreadBlockAllocations[ALLOCATIONMANAGEMENTTHREADNUMBER];

        for (int i = 0; i < ALLOCATIONMANAGEMENTTHREADNUMBER; ++i) {
            pthreadBlockAllocations[i].targs = args;
            pthreadBlockAllocations[i].start = i * (BLOCKNUMBER / ALLOCATIONMANAGEMENTTHREADNUMBER);
            pthreadBlockAllocations[i].end = (i + 1) * (BLOCKNUMBER / ALLOCATIONMANAGEMENTTHREADNUMBER);

            pthread_create(&threads[i], NULL, blockAllocationThr, &pthreadBlockAllocations[i]);
        }

        for (int i = 0; i < ALLOCATIONMANAGEMENTTHREADNUMBER; ++i) {
            pthread_join(threads[i], NULL);
        }
    }
    return NULL;

    // while(1)
    // {
    //     while (args->pMMUOnTransfer->syncFlag>0 && !args->should_exit) {
    //     }
    //     if(args->should_exit)break;
        
    //     //allocating calculate
    //     args->pmemoryManagement->allocateMem(args->pMMUOnTransfer);
        
    //     //after finishing, notify GPU to continue
    //     args->pMMUOnTransfer -> syncFlag = 1;
    // }
    // pthread_exit(NULL);
}

//each thread deal with block allocations
void* blockAllocationThr(void* arg)
{
    threadBlockAllocations* data = (threadBlockAllocations*)arg;
    
    while (1) {
        for (int i = data->start; i < data->end; ++i) {
            //chek if there are allocation
            if (data->targs->pMMUOnTransfer->syncFlag[i] == 0){
                //allocating calculate
                data -> targs -> pmemoryManagement->allocateMem(data->targs->pMMUOnTransfer, i);

                //after finishing, notify GPU to continue
                data->targs -> pMMUOnTransfer -> syncFlag[i] = 1;
            }

            //chek if there are free
            if (data->targs->pMMUOnTransfer->syncFlag[i] == 2){
                //allocating calculate
                data -> targs -> pmemoryManagement->freeMem(data->targs->pMMUOnTransfer, i);
                
                //after finishing, notify GPU to continue
                data->targs -> pMMUOnTransfer -> syncFlag[i] = 1;
            }
        }
        if(data->targs -> should_exit) break;
    }

    pthread_exit(NULL);
}

//after launch kernel
void aLaunchKernel(MemoryManagement* memoryManagement, struct MMUOnTransfer *pMMUOnTransfer, hipStream_t stream)
{
    //thread launch
    pthread_t thread_id;

    // Initialize thread arguments
    thread_args args = { .pmemoryManagement = memoryManagement, .pMMUOnTransfer = pMMUOnTransfer, .should_exit = 0}; 

    // Create the thread
    pthread_create(&thread_id, NULL, threadAllocation, &args);
    // if (pthread_create(&thread_id, NULL, threadAllocation, &args)) {
    //     fprintf(stderr, "Error creating thread\n");
    // }

    // Wait for the kernel in this stream to complete
    hipError_t error = hipStreamSynchronize(stream);
    
    // Signal the thread to exit
    args.should_exit = 1;

    // Wait for the thread to finish
    pthread_join(thread_id, NULL);
    // if (pthread_join(thread_id, NULL)) {
    //     fprintf(stderr, "Error joining thread\n");
    // }

    hipStreamDestroy(stream);
}

//before launch kernel, init it
void initAllocationStru(MemoryManagement* memoryManagement, struct MMUOnTransfer *pMMUOnTransfer)
{
    unsigned int threadNumber = BLOCKNUMBER * BLOCKSIZE;
    pMMUOnTransfer -> bitmapStartAddress = memoryManagement->getBitmapStartAddress();
    pMMUOnTransfer -> linkedListStartAddress = memoryManagement->getLinkedListStartAddress();
    for(int i = 0; i < BLOCKNUMBER; i++)
    {
        pMMUOnTransfer -> syncFlag[i] = 1;
    }
    
    for(int i = 0; i < threadNumber; i++)
    {
        pMMUOnTransfer -> sizeAllocate[i] = 0;
        pMMUOnTransfer -> addressAllocate[i] = NULL;
        pMMUOnTransfer -> addressFree[i] = NULL;
    }
}
/* -- End -- allocation function*/

/* -- allocation in GPU*/
//allocation on Each thread
__device__ void* allocateThr(size_t allocateSize, struct MMUOnTransfer* pMMUOnTransfer)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    pMMUOnTransfer -> sizeAllocate[tid + bid * BLOCKSIZE] = allocateSize;
    
    // Ensure all threads have written their allocating size to transfer structure
    __syncthreads();
    
    //after all threads finish send the allocating size to transfer structure,
    //thread 0 sets the flag to zero, to notify CPU to calculate the allocation.
    if(tid == 0)
    {
        // atomicSub(&(pMMUOnTransfer -> syncFlag[bid]), 1);
        //allocation tag
        pMMUOnTransfer -> syncFlag[bid] = 0;
    }
    
    // Ensure threads o have set the flag successfully
    __syncthreads();
    // Wait for the host to finish allocation and set the continue flag
    while (atomicAdd(&(pMMUOnTransfer -> syncFlag[bid]), 0) == 0) 
    {
    }

    //read the allocation adderess
    return pMMUOnTransfer -> addressAllocate[tid + bid * BLOCKSIZE];
}

__device__ void freeThr(void* freeAddress, struct MMUOnTransfer* pMMUOnTransfer)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    pMMUOnTransfer -> addressFree[tid + bid * BLOCKSIZE] = freeAddress;
    
    // Ensure all threads have written their allocating size to transfer structure
    __syncthreads();
    
    //after all threads finish send the allocating size to transfer structure,
    //thread 0 sets the flag to zero, to notify CPU to calculate the allocation.
    if(tid == 0)
    {
        // atomicSub(&(pMMUOnTransfer -> syncFlag[bid]), 1);
        //free tag
        pMMUOnTransfer -> syncFlag[bid] = 2;
    }
    
    // Ensure threads o have set the flag successfully
    __syncthreads();
    // Wait for the host to finish allocation and set the continue flag
    while (atomicAdd(&(pMMUOnTransfer -> syncFlag[bid]), 0) == 0) 
    {
    }

}
/* -- End -- allocation in GPU*/