#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "MMUOnCPU.hpp"
#include <hip/hip_runtime.h>

#include <time.h>


/* -- test for version 1.2 --*/
__global__ void waitForHostAndContinue(struct MMUOnTransfer* pMMUOnTransfer)
{
    // int n = 99321021;
    // int count = 0;
    // for(int i = 0; i < n; i++)
    // {
    //     if(n % i == 0)
    //     {
    //         count++;
    //     }
    // }
    // pMMUOnTransfer -> sizeAllocate[BLOCKSIZE*blockIdx.x + threadIdx.x] = count; 
    //allocate call
    int tid =  threadIdx.x;
    int* a = (int*)allocateThr(8192, pMMUOnTransfer);
    
    // *a = tid + 20;

    // if(tid == 0)printf("a\n");

    // int* b = (int*)allocateThr(10, pMMUOnTransfer);
    // *b = tid + 30;
    // freeThr(a, pMMUOnTransfer);
}

int main() 
{
    /* -- init meta -- */
    // init MemoryManagement
    MemoryManagement* memoryManagement[ALLOCATIONMANAGEMENTTHREADNUMBER];
    struct MMUOnTransfer* pMMUOnTransfer;
    hipError_t cudaStatus = hipMallocManaged(&pMMUOnTransfer, sizeof(struct MMUOnTransfer));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    //init MMUOnTransfer
    initAllocationStru(memoryManagement, pMMUOnTransfer);
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch the kernel
    waitForHostAndContinue<<<BLOCKNUMBER, BLOCKSIZE, 0, stream>>>(pMMUOnTransfer);

    aLaunchKernel(memoryManagement, pMMUOnTransfer, stream);

    // Record the stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); // Wait for the kernel to complete
    printf("pMMUOnTransfer : %d\n", pMMUOnTransfer -> sizeAllocate[0]);
    
    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time:  %f  milliseconds\n", milliseconds);
    
    hipFree(pMMUOnTransfer);
}
