#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "MMUOnCPU.hpp"
#include <hip/hip_runtime.h>
#include <time.h>


/* 
generate a new random order with fixed sizes

compiler command
nvcc -rdc=true testshbench.cu MMUOnCPU.cu -lpthread
*/

#define ALL_FREE_TIMES 16000
#define PORTION_PER_THREAD (PORTIONS_PER_BLOCK)

typedef struct Node {
    int data;
    struct Node* next;
} Node;

// Function prototypes
Node* createNode(int data);

// Function to check if a value exists in the list
int existsInList(Node* head, int data);

// Function to insert a new unique random number into the list
int insertUniqueRandom(Node** head);

// Function to delete a random node from the list
int deleteRandom(Node** head, int length);

void generateSeq(bool* isAllocate, int* freeBlock);

__global__ void testRandomly(bool* d_isAllocate, int* d_freeBlock, struct MMUOnTransfer* pMMUOnTransfer)
{
    //init memory pool used stack
    StackUsedPool s;
    initStackUsedPool(&s);

    int* allocationPointers[PORTION_PER_THREAD];
    for(int i = 0; i < ALL_FREE_TIMES; i++)
    {
    //     // if(threadIdx.x == 0)
    //     // {
    //     //     printf("AA-");
    //     // }
        
        if(d_isAllocate[i] == 1)
        {
            allocationPointers[d_freeBlock[i]] = (int*)allocateThr(8192, pMMUOnTransfer);
        }else
        {
            freeThr(allocationPointers[d_freeBlock[i]], pMMUOnTransfer);
        }
        if(threadIdx.x == 0)
        {
            printf("A\n");
        }
    }
}

int main() 
{
    
    /* -- init meta -- */
    // init MemoryManagement
    MemoryManagement* memoryManagement[ALLOCATIONMANAGEMENTTHREADNUMBER];
    struct MMUOnTransfer* pMMUOnTransfer;
    pthread_t thread_id;
    thread_args args;
    int should_exit = 0;
    hipError_t cudaStatus = hipMallocManaged(&pMMUOnTransfer, sizeof(struct MMUOnTransfer));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    //init MMUOnTransfer
    initAllocationStru(memoryManagement, pMMUOnTransfer, &thread_id, &args, &should_exit);

    hipStream_t stream;
    hipStreamCreate(&stream);

    //generate a allocate or free sequence in a random order
    bool isAllocate[ALL_FREE_TIMES];
    int freeBlock[ALL_FREE_TIMES];

    //device varaiables
    bool *d_isAllocate;
    int *d_freeBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory on the device
    hipMalloc(&d_isAllocate, ALL_FREE_TIMES * sizeof(bool));
    hipMalloc(&d_freeBlock, ALL_FREE_TIMES * sizeof(int));

    //init allocatio sequence list
    generateSeq(isAllocate, freeBlock);
    // for(int i =0; i<ALL_FREE_TIMES; i++)
    // {
    //     printf("%d\n",isAllocate[i]);
    // }

    // Copy the host array to the device
    hipMemcpy(d_isAllocate, isAllocate, ALL_FREE_TIMES * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_freeBlock, freeBlock, ALL_FREE_TIMES * sizeof(int), hipMemcpyHostToDevice);

    // Record the start event
    hipEventRecord(start, 0);

    // Launch the kernel
    testRandomly<<<BLOCKNUMBER, BLOCKSIZE, 0, stream>>>(d_isAllocate, d_freeBlock, pMMUOnTransfer);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop); 
    aLaunchKernel(&args, stream);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time:  %f  milliseconds\n", milliseconds);
    // std::cout << "Kernel execution time: " << milliseconds << " milliseconds\n";

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device memory
    hipFree(d_isAllocate);
    hipFree(d_freeBlock);
    
    return 0;



    // // Launch the kernel
    // waitForHostAndContinue<<<BLOCKNUMBER, BLOCKSIZE>>>();
}

//implementation
Node* createNode(int data) 
{
    Node* newNode = (Node*)malloc(sizeof(Node));
    if (newNode == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
    newNode->data = data;
    newNode->next = NULL;
    return newNode;
}

int existsInList(Node* head, int data) 
{
    Node* current = head;
    while (current != NULL) {
        if (current->data == data) {
            return 1; // Data found
        }
        current = current->next;
    }
    return 0; // Data not found
}

int insertUniqueRandom(Node** head) 
{
    while(1)
    {
        int randomData = rand() % PORTION_PER_THREAD; // Random number between 0 and PORTION_PER_THREAD
        if (!existsInList(*head, randomData)) {
            Node* newNode = createNode(randomData);
            newNode->next = *head;
            *head = newNode;
            // printf("insert %d\n", randomData);
            return randomData;
        } 
    }
}

int deleteRandom(Node** head, int length) 
{
    int randomIndex = rand() % length;
    Node* prev = NULL;
    Node* current = *head;

    if (randomIndex == 0) { // Delete the head
        *head = current->next;
        int valueD = current -> data;
        free(current);
        // printf("Deleted %d \n", valueD);
        return valueD;
    }

    for (int i = 0; i < randomIndex; i++) {
        prev = current;
        current = current->next;
    }

    prev->next = current->next;
    int valueD = current -> data;
    free(current);
    // printf("Deleted %d\n", valueD);
    return valueD;
}

void generateSeq(bool* isAllocate, int* freeBlock)
{
    srand(time(NULL));
    Node* head = NULL;
    int nodeLength = 0;

    for(int i = 0; i < ALL_FREE_TIMES; i++)
    {
        //if the allocated list is empty, allocate one.
        if(nodeLength == 0)
        {
            freeBlock[i] = insertUniqueRandom(&head);
            isAllocate[i] = 1;
            nodeLength ++;
            continue;
        }

        //if the allocated list is full, delete one.
        if(nodeLength == PORTION_PER_THREAD)
        {
            isAllocate[i] = 0;
            freeBlock[i] = deleteRandom(&head, nodeLength);
            nodeLength --;
            continue;
        }

        //allocate or free randomly
        int isallocate = rand() % 2;
        //if allocate
        if(isallocate == 1)
        {
            freeBlock[i] = insertUniqueRandom(&head);
            isAllocate[i] = 1;
            nodeLength ++;
        }else
        {
            isAllocate[i] = 0;
            freeBlock[i] = deleteRandom(&head, nodeLength);
            nodeLength --;
        }
    }
}
