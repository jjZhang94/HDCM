#include "hip/hip_runtime.h"
#include "MMUOnCPU.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#include <hiprand/hiprand_kernel.h>

#include <unistd.h>

/* -- version 1.3*/
// Memory pool structure
__device__ char memoryPool[POOL_SIZE];
__device__ int allocationMapGPU[TOTAL_PORTIONS]; // 0 = free, 1 = allocated
/* -- End -- version 1.3*/

/* -- version 1.3.1*/
// Global variable for CURAND states
__device__ hiprandState curandStates[TOTAL_PORTIONS];
/* -- End -- version 1.3.1*/

/* -- Implementation of memory management with linked lists -- */
LinkedListManagement::LinkedListManagement(void * startAddressIn)
{
    //init start address
    startAddress = startAddressIn;

    //create a init blanket Node
    MemorySegmentNode* newMemorySegmentNode = (MemorySegmentNode *)malloc(sizeof(MemorySegmentNode));
    if (newMemorySegmentNode == NULL) {
        fprintf(stderr, "Error allocating memory for list node\n");
        exit(EXIT_FAILURE);
    }
    newMemorySegmentNode->length = MAX_LINKEDLIST_MEM_SIZE;
    newMemorySegmentNode->offsetAddress = 0;
    newMemorySegmentNode->next = NULL;
    newMemorySegmentNode->isHole = true;
    
    ListHead = newMemorySegmentNode;

}

LinkedListManagement::~LinkedListManagement()
{
    //free the entire list of MemorySegmentNode
    MemorySegmentNode* currentNode = ListHead;
    MemorySegmentNode* nextNode;
    while(currentNode != NULL){
        nextNode = currentNode -> next;
        free(currentNode);
        currentNode = nextNode;
    }
    ListHead = NULL;

}

MemorySegmentNode* LinkedListManagement::findSuitableMem(unsigned int sizeAllocate)
{
    //return the current node when finding a suitable node
    MemorySegmentNode* returnNode = NULL;
    MemorySegmentNode* currentNode = ListHead;
    while (currentNode != NULL) {
        if((currentNode -> isHole == true) && (currentNode ->length >= sizeAllocate))
        {
            return currentNode;
        }
        currentNode = currentNode->next;
    }
    return returnNode;
}

void* LinkedListManagement::allocateMem(unsigned int sizeAllocate)
{
    void* returnAddress = NULL;
    MemorySegmentNode* insertNode = NULL;
    //find a suitable position to allocate and return the pointer to the last node.
    insertNode = findSuitableMem(sizeAllocate);
    
    //if cannot find a suitable memory block
    if (insertNode == NULL){
        fprintf(stderr, "Error allocating memory for memory\n");
        exit(EXIT_FAILURE);
    }

    /*------Update memory management nodes--------*/
    returnAddress = (void*)((char*)startAddress + insertNode -> offsetAddress);
    //If the allocated length is equal to memory block existed, just chage the status of memory management node
    //isHole to false.
    if(insertNode->length == sizeAllocate)
    {
        insertNode -> isHole = false;
        return returnAddress;
    }

    //create a new Node
    MemorySegmentNode* newMemorySegmentNode = (MemorySegmentNode *)malloc(sizeof(MemorySegmentNode));
    if (newMemorySegmentNode == NULL) {
        fprintf(stderr, "Error allocating memory for list node\n");
        exit(EXIT_FAILURE);
    }

    newMemorySegmentNode -> length = insertNode -> length - sizeAllocate;
    newMemorySegmentNode -> offsetAddress = insertNode -> offsetAddress + sizeAllocate;
    newMemorySegmentNode -> next = insertNode -> next;
    newMemorySegmentNode -> isHole = true;

    //upadate original nodes
    insertNode -> isHole = false;
    insertNode -> length = sizeAllocate;
    insertNode -> next = newMemorySegmentNode;
    
    return returnAddress;
}

void LinkedListManagement::freeMem(void* addressIn)
{
    /*--the lack of check the avaliable scope of addressIn.-- */
    unsigned int currentAddressOffset = (unsigned int)((char*)addressIn - (char*)startAddress);
    //Find the corresponding node
    MemorySegmentNode* currentNode = ListHead;
    MemorySegmentNode* lastNode = NULL;
    MemorySegmentNode* nextNode = NULL;
    while (currentNode != NULL) {
        if (currentNode->offsetAddress == currentAddressOffset) {
            break;
        }
        lastNode = currentNode;
        currentNode = currentNode->next;
    }

    nextNode = currentNode->next;
    //check the results
    // if (currentNode == NULL){
    //     /*---if cannot find it, return an error----*/
    //     return;
    // }

    /* -- merge the neighbor idle memory blocks and clean the unnecessary management nodes*/
    if(lastNode == NULL)
    {
        if(nextNode == NULL)
        {
            currentNode -> isHole = true;
        }else
        {
            if(nextNode ->isHole == false){
                currentNode -> isHole = true;
            }else{
                currentNode -> isHole = true;
                currentNode -> length = nextNode -> length + currentNode -> length;
                currentNode -> next = nextNode -> next;
                free(nextNode);
            }
        }
        return;
    }

    if(nextNode == NULL)
    {
        if(lastNode -> isHole == true)
        {
            lastNode -> next = nextNode;
            lastNode -> length = lastNode -> length + currentNode -> length;
            free(currentNode);
        }else{
            currentNode -> isHole = true;
        }
        return;
    }

    if(lastNode-> isHole == true)
    {
        lastNode -> next = nextNode;
        lastNode -> length = lastNode -> length + currentNode -> length;
        free(currentNode);
        if(nextNode -> isHole == true)
        {
            lastNode -> next = nextNode -> next;
            lastNode -> length = lastNode -> length + nextNode -> length;
            free(nextNode);
        }
        //else if nextNode's isHole is false, do nothing
    }else
    {
        if(nextNode -> isHole == true)
        {
            currentNode -> isHole = true;
            currentNode -> length = nextNode -> length + currentNode -> length;
            currentNode -> next = nextNode -> next;
            free(nextNode);
        }else{
            currentNode -> isHole = true;
        }
    }

}
/* -- End -- Implementation of memory management with linked lists -- */


/* -- Implementation of memory management with bitmaps -- */
BitmapManagement::BitmapManagement(void * startAddressIn)
{
    //init start address
    startAddress = startAddressIn;

    //init bitmap
    initializeBitmap();

    //init hash tables
    initializeHashTable();
}

BitmapManagement::~BitmapManagement()
{
    freeHashTable();
}

void BitmapManagement::initializeBitmap()
{
    memset(bitmap, 0, sizeof(bitmap));
    return;
}

void BitmapManagement::setBitMapTrue(unsigned int blockIndex)
{
    unsigned int byteIndex = blockIndex / 8;
    unsigned int bitIndex = blockIndex % 8;
    unsigned char mask = 1 << bitIndex;
    bitmap[byteIndex] |= mask;
    return;
}

void BitmapManagement::setBitsMapTrue(unsigned int size, unsigned int startBlockIndex)
{
    unsigned int endBlockIndex = startBlockIndex + size;
    unsigned int startByteIndex = startBlockIndex / 8;
    unsigned int endByteIndex = (endBlockIndex - 1) / 8;

    if (startByteIndex == endByteIndex)
    {
        // If size is no larger than 8 and within a single byte
        unsigned int startBitIndex = startBlockIndex % 8;
        unsigned int endBitIndex = endBlockIndex % 8;
        unsigned char mask = ((0xFF >> (8 - endBitIndex)) & (0xFF << startBitIndex));

        bitmap[startByteIndex] |= mask;
        return;
    }

    for (unsigned int i = startByteIndex; i <= endByteIndex; i++) {
        unsigned char mask;

        if (i == startByteIndex) {
            // Create mask for the first byte
            unsigned int startBitIndex = startBlockIndex % 8;
            mask = (0xFF << startBitIndex);
        } else if (i == endByteIndex) {
            // Create mask for the last byte
            unsigned int endBitIndex = endBlockIndex % 8;
            if(endBitIndex == 0)
                endBitIndex = endBitIndex + 8;
            mask = (0xFF >> (8 - endBitIndex));
        } else {
            // Set all bits in the intermediate bytes
            mask = 0xFF;
        }

        bitmap[i] |= mask;
    }
    return;
}

void BitmapManagement::clearBit(unsigned int blockIndex)
{
    unsigned int byteIndex = blockIndex / 8;
    unsigned int bitIndex = blockIndex % 8;
    unsigned char mask = ~(1 << bitIndex);
    bitmap[byteIndex] &= mask;
}

void BitmapManagement::clearBits(unsigned int size, unsigned int startBlockIndex)
{
    unsigned int endBlockIndex = startBlockIndex + size;
    unsigned int startByteIndex = startBlockIndex / 8;
    unsigned int endByteIndex = (endBlockIndex - 1) / 8;

    if (startByteIndex == endByteIndex)
    {
        // If size is no larger than 8 and within a single byte
        unsigned int startBitIndex = startBlockIndex % 8;
        unsigned int endBitIndex = endBlockIndex % 8;
        unsigned char mask = ~((0xFF >> (8 - endBitIndex)) & (0xFF << startBitIndex));

        bitmap[startByteIndex] &= mask;
        return;
    }

    // Handle multiple bytes
    for (unsigned int i = startByteIndex; i <= endByteIndex; i++) 
    {
        unsigned char mask;
        if (i == startByteIndex) {
            // Create mask for the first byte
            unsigned int startBitIndex = startBlockIndex % 8;
            mask = ~(0xFF << startBitIndex);
        } else if (i == endByteIndex) {
            // Create mask for the last byte
            unsigned int endBitIndex = endBlockIndex % 8;
            if(endBitIndex == 0)
                endBitIndex = endBitIndex + 8;
            mask = ~(0xFF >> (8 - endBitIndex));
        } else {
            // Clear all bits in the intermediate bytes
            mask = 0x00;
        }

        bitmap[i] &= mask;
    }
    return;
}

bool BitmapManagement::isBitSet(unsigned int blockIndex)
{
    unsigned int byteIndex = blockIndex / 8;
    unsigned int bitIndex = blockIndex % 8;
    unsigned char mask = 1 << bitIndex;
    return (bool)(bitmap[byteIndex] & mask) != 0;    
}

bool BitmapManagement::isZeroBit(unsigned char byte, unsigned int position) 
{
    return (byte & (1 << position)) == 0;
}

int BitmapManagement::isByteFree(unsigned int byteIndex)
{
    // Iterate through each bit in the char value
    for (int i = 0; i < 8; ++i) {
        // Check if the ith bit is 0 using bitwise AND
        if ((bitmap[byteIndex] & (1 << i)) == 0) {
            return i; // Return the position of the first 0 bit
        }
    }
    return -1; // Return -1 if all bits are 1
}

unsigned int BitmapManagement::findPage()
{
    //Simple sequential search
    for(unsigned int i =0; i < BITMAP_SIZE; i++)
    {
        int index = isByteFree(i);
        if (index >= 0)
        {
            return i*8+index; // Return the index of the allocated block
        }
    }
    return 0;
}

unsigned int BitmapManagement::findPages(unsigned int pageSize)
{
    //find continuous pages
    //Simple sequential search
    unsigned int count = 0; // Count of continuous zeros
    for (unsigned int i = 0; i < PAGE_NUMBER; ++i) {
        if (isZeroBit(bitmap[i / 8], i % 8)) {
            count++;
            if (count == pageSize) {
                return i - pageSize + 1; // Return the starting index of the sequence
            }
        } else {
            count = 0; // Reset count if a one is found
        }
    }
    return 0; // Return 0 if not found
}

void* BitmapManagement::allocateMem(unsigned int sizeAllocate)
{
    //get the size of continuous pages
    unsigned int pages = ceil(float(sizeAllocate) / PAGE_SIZE);
    //if just need only one page, just find a idle block
    if (pages == 1)
    {  
        unsigned int targetBlockIndex = findPage();
        setBitMapTrue(targetBlockIndex);
        insertHashTable(targetBlockIndex, pages);
        return (void*)((char*)startAddress + targetBlockIndex*PAGE_SIZE);
    }

    //more than one pages
    unsigned int startBlockIndex = findPages(pages);
    // printf("startBlockIndex %d\n", startBlockIndex);
    setBitsMapTrue(pages, startBlockIndex);

    //update block-size table
    insertHashTable(startBlockIndex, pages);

    return (void*)((char*)startAddress + startBlockIndex*PAGE_SIZE);

}

void BitmapManagement::freeMem(void* addressIn)
{
    //get the size of blockID
    unsigned int offsetAddress = (unsigned int)((char*)addressIn - (char*)startAddress);
    unsigned int blockID = offsetAddress / PAGE_SIZE;

    //search the blockID and its size
    unsigned pageNumber = findHashTable(blockID);
    if(pageNumber == 0) return;
    //if just free only one page, just clear the block
    if(pageNumber == 1)
    {
        clearBit(pageNumber);  
    }else
    {
        // printf("clearn %d, %d\n", pageNumber, blockID);
        clearBits(pageNumber, blockID);
    }
    
    deleteHashTable(blockID);
    return;
      
}

unsigned int BitmapManagement::hash(unsigned int key)
{
    key = ((key >> 16) ^ key) * 0x45d9f3b;
    key = ((key >> 16) ^ key) * 0x45d9f3b;
    key = (key >> 16) ^ key;
    return key & (HASHTABLE_BITMAP_BLOCKIDSIZE_SIZE - 1); // TABLE_SIZE is 4096
}

void BitmapManagement::initializeHashTable()
{
    for (int i = 0; i < HASHTABLE_BITMAP_BLOCKIDSIZE_SIZE; ++i) {
        buckets[i] = NULL;
    }
}

void BitmapManagement::insertHashTable(unsigned int key, unsigned int value)
{
    unsigned int idx = hash(key);
    BlockIDSizeNode* newNode = (BlockIDSizeNode*)malloc(sizeof(BlockIDSizeNode));

    newNode->pair.blockID = key;
    newNode->pair.size = value;

    // Insert at the beginning of the chain (bucket)
    newNode->next = buckets[idx];
    buckets[idx] = newNode;
}

unsigned int BitmapManagement::findHashTable(unsigned int key)
{
    unsigned int idx = hash(key);
    unsigned int returnValue;
    BlockIDSizeNode* current = buckets[idx];
    while (current != NULL) {
        if (current->pair.blockID == key)
        {
            returnValue = current->pair.size;
            return returnValue;
        }
        current = current->next;
    }
    return 0;
}

void BitmapManagement::deleteHashTable(unsigned int key)
{
    unsigned int idx = hash(key);
    BlockIDSizeNode *current = buckets[idx];
    BlockIDSizeNode *prev = NULL;
    while (current != NULL) {
        if (current->pair.blockID == key) {
            if (prev == NULL) {
                buckets[idx] = current->next;
            } else {
                prev->next = current->next;
            }
            free(current);
            return;
        }
        prev = current;
        current = current->next;
    }
}

void BitmapManagement::freeHashTable()
{
    for (int i = 0; i < HASHTABLE_BITMAP_BLOCKIDSIZE_SIZE; ++i) {
        BlockIDSizeNode* current = buckets[i];
        while (current != NULL) {
            BlockIDSizeNode* temp = current;
            current = current->next;
            free(temp);
        }
    }
}
/* -- End -- Implementation of memory management with bitmaps -- */


/* -- Implementation of memory management -- */
MemoryManagement::MemoryManagement()
{
    //init startAddress

    // bitmapStartAddress = (void*)malloc(MAX_BITMAP_MEM_SIZE);
    // linkedListStartAddress = (void*)malloc(MAX_LINKEDLIST_MEM_SIZE);

    long long size = 1024LL*1024LL*1024LL * 16;
    // size = size * 4;
    hipMallocManaged(&bitmapStartAddress, size);
    size = 1024*1024*1024;
    hipMallocManaged(&linkedListStartAddress, size);

    //init class BitmapManagement and LinkedListManagement
    bitmapManagement = new BitmapManagement(bitmapStartAddress);
    linkedListManagement = new LinkedListManagement(linkedListStartAddress);
}

MemoryManagement::~MemoryManagement()
{
    delete bitmapManagement;
    delete linkedListManagement;

    // free(bitmapStartAddress);
    // free(linkedListStartAddress);

    hipFree(bitmapStartAddress);
    hipFree(linkedListStartAddress);
}

void MemoryManagement::allocateMem(struct MMUOnTransfer* pMMUOnTransfer)
{
    // printf("SSDD %d\n", bid);
    //linkedListManagement
    
    if(pMMUOnTransfer -> sizeAllocate  * 8 > THRESHOLD_MEMSIZE)
    {
        pMMUOnTransfer -> addressAllocate = bitmapManagement->allocateMem(pMMUOnTransfer -> sizeAllocate);
    }else
    {
        pMMUOnTransfer -> addressAllocate = bitmapManagement->allocateMem(pMMUOnTransfer -> sizeAllocate);
    }
    // printf("allocate %d address %p, siez %d\n", pMMUOnTransfer -> addressAllocate, pMMUOnTransfer -> sizeAllocate);
    // sleep(0.3);
}

void MemoryManagement::freeMem(struct MMUOnTransfer* pMMUOnTransfer)
{
    void* bitmapEndAddress = (void*)((char*)(bitmapStartAddress) + MAX_BITMAP_MEM_SIZE);
    // printf("free address %p\n", pMMUOnTransfer -> addressFree);
    if(pMMUOnTransfer -> addressFree == NULL)
    {
        printf("NULL\n");
    }
    bitmapManagement->freeMem(pMMUOnTransfer -> addressFree);

    // if((pMMUOnTransfer -> addressFree[bid]) >= bitmapStartAddress && (pMMUOnTransfer -> addressFree[bid]) < bitmapEndAddress)
    // {
    //     bitmapManagement->freeMem(pMMUOnTransfer -> addressFree[bid]);
    // }else
    // {
    //     bitmapManagement->freeMem(pMMUOnTransfer -> addressFree[bid]);
    // }
}

void* MemoryManagement::getBitmapStartAddress()
{
    return bitmapStartAddress;
}

void* MemoryManagement::getLinkedListStartAddress()
{
    return linkedListStartAddress;
}
/* -- End -- Implementation of memory management -- */

/* -- allocation function*/
//launch threads to deal with blocks allocation and free
void* threadAllocation(void* arg)
{
    thread_args* args = (thread_args*)arg;
    
    //create threads for block allocations
    if(BLOCKNUMBER < ALLOCATIONMANAGEMENTTHREADNUMBER)
    {
        // if the number of threads smaller than blocknumber
        // Create and start threads
        pthread_t threads[BLOCKNUMBER];
        threadBlockAllocations pthreadBlockAllocations[BLOCKNUMBER];
        for(int i = 0; i < BLOCKNUMBER; ++i) 
        {
            pthreadBlockAllocations[i].pMMUOnTransfer = args -> pMMUOnTransfer;
            pthreadBlockAllocations[i].should_exit = args -> should_exit;
            pthreadBlockAllocations[i].pmemoryManagement = args -> pmemoryManagement[i];
            pthreadBlockAllocations[i].start = i;
            pthreadBlockAllocations[i].end = i + 1;

            pthread_create(&threads[i], NULL, blockAllocationThr, &pthreadBlockAllocations[i]);
        }

        for (int i = 0; i < BLOCKNUMBER; ++i) {
            pthread_join(threads[i], NULL);
        }
    }else{
        pthread_t threads[ALLOCATIONMANAGEMENTTHREADNUMBER];
        threadBlockAllocations pthreadBlockAllocations[ALLOCATIONMANAGEMENTTHREADNUMBER];

        for (int i = 0; i < ALLOCATIONMANAGEMENTTHREADNUMBER; ++i) {
            pthreadBlockAllocations[i].pMMUOnTransfer = args -> pMMUOnTransfer;
            pthreadBlockAllocations[i].should_exit = args -> should_exit;
            pthreadBlockAllocations[i].pmemoryManagement = args -> pmemoryManagement[i];
            pthreadBlockAllocations[i].start = i * (BLOCKNUMBER / ALLOCATIONMANAGEMENTTHREADNUMBER);
            pthreadBlockAllocations[i].end = (i + 1) * (BLOCKNUMBER / ALLOCATIONMANAGEMENTTHREADNUMBER);
            if(i == (ALLOCATIONMANAGEMENTTHREADNUMBER - 1))
            {
                pthreadBlockAllocations[i].end = BLOCKNUMBER;
            }
            // printf("start %d. end:%d\n", pthreadBlockAllocations[i].start, pthreadBlockAllocations[i].end);
            pthread_create(&threads[i], NULL, blockAllocationThr, &pthreadBlockAllocations[i]);
        }

        for (int i = 0; i < ALLOCATIONMANAGEMENTTHREADNUMBER; ++i) {
            pthread_join(threads[i], NULL);
        }
    }
    return NULL;

    // while(1)
    // {
    //     while (args->pMMUOnTransfer->syncFlag>0 && !args->should_exit) {
    //     }
    //     if(args->should_exit)break;
        
    //     //allocating calculate
    //     args->pmemoryManagement->allocateMem(args->pMMUOnTransfer);
        
    //     //after finishing, notify GPU to continue
    //     args->pMMUOnTransfer -> syncFlag = 1;
    // }
    // pthread_exit(NULL);
}

//each thread deal with block allocations
void* blockAllocationThr(void* arg)
{
    threadBlockAllocations* data = (threadBlockAllocations*)arg;
    
    while (true) {
        for (int i = data->start; i < data->end; ++i) {
            //chek if there are allocation
            if ((data->pMMUOnTransfer[i])->syncFlag == 0){
                //allocating calculate
                // printf("bb\n");
                data -> pmemoryManagement->allocateMem(data->pMMUOnTransfer[i]);
                // printf("allocate %d %p\n", i, (data->pMMUOnTransfer[i])->addressAllocate);
                // Perform some calculation...
                // printf("aa\n");
                //after finishing, notify GPU to continue
                (data -> pMMUOnTransfer[i]) -> syncFlag = 1;
            }
            
            //chek if there are free
            if ((data->pMMUOnTransfer[i])->syncFlag == 2){
                //allocating calculate
                
                data -> pmemoryManagement->freeMem(data->pMMUOnTransfer[i]);
                // Perform some calculation...
                
                //after finishing, notify GPU to continue
                (data -> pMMUOnTransfer[i]) -> syncFlag = 1;

            }
        }

        //when kernel finishes, checking is over
        if(*(data -> should_exit))
        {
            break;
        }
    }

    pthread_exit(NULL);
}

//after launch kernel
void aLaunchKernel(thread_args* args, hipStream_t stream)
{
    // //thread launch
    // pthread_t thread_id;

    // // Initialize thread arguments
    // thread_args args;
    //  = { .pMMUOnTransfer = pMMUOnTransfer}; 
    // int ii = 0;
    // args.should_exit = &ii;
    // for(int i = 0; i < ALLOCATIONMANAGEMENTTHREADNUMBER; i++)
    // {
    //     args.pmemoryManagement[i] = memoryManagement[i];
    // }

    // // Create the thread
    // pthread_create(&thread_id, NULL, threadAllocation, &args);
    // if (pthread_create(&thread_id, NULL, threadAllocation, &args)) {
    //     fprintf(stderr, "Error creating thread\n");
    // }
    
    // Wait for the kernel in this stream to complete
    hipError_t error = hipStreamSynchronize(stream);
    
    // Signal the thread to exit
    *(args -> should_exit) = 1;

    // Wait for the thread to finish
    // pthread_join(thread_id, NULL);

    // if (pthread_join(thread_id, NULL)) {
    //     fprintf(stderr, "Error joining thread\n");
    // }

    hipStreamDestroy(stream);
}

//before launch kernel, init it
void initAllocationStru(MemoryManagement* memoryManagement[], struct MMUOnTransfer **pMMUOnTransfer, pthread_t* thread_id, thread_args* args, int* should_exit)
{
    // pMMUOnTransfer -> bitmapStartAddress = memoryManagement->getBitmapStartAddress();
    // pMMUOnTransfer -> linkedListStartAddress = memoryManagement->getLinkedListStartAddress();
    
    //create MemoryManagement
    for(int i = 0; i < ALLOCATIONMANAGEMENTTHREADNUMBER; i++)
    {
        memoryManagement[i] = new MemoryManagement();
    }

    for(int i = 0; i < BLOCKNUMBER; i++)
    {
        pMMUOnTransfer[i] -> sizeAllocate = 0;
        pMMUOnTransfer[i] -> syncFlag = 1;
        pMMUOnTransfer[i] -> addressAllocate = NULL;
        pMMUOnTransfer[i] -> addressFree = NULL;
    }

    // Initialize thread arguments
    args ->  pMMUOnTransfer = pMMUOnTransfer; 
    args -> should_exit = should_exit;
    for(int i = 0; i < ALLOCATIONMANAGEMENTTHREADNUMBER; i++)
    {
        args -> pmemoryManagement[i] = memoryManagement[i];
    }

    if (pthread_create(thread_id, NULL, threadAllocation, args)) {
        fprintf(stderr, "Error creating thread\n");
    }
}
/* -- End -- allocation function*/

/* -- allocation in GPU*/

/* -- End -- allocation in GPU*/

/* Function in Version 1.3*/
//The host initializes the GPU memory pool.
void initMemoryPoolOnHost()
{
    initMemoryPool<<<BLOCKNUMBER, PORTIONS_PER_BLOCK>>>();
    hipDeviceSynchronize();
}

__global__ void initMemoryPool()
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < TOTAL_PORTIONS) {
        allocationMapGPU[idx] = 0; // Mark as free

        /* Function in version 1.3.1*/
        //randomly page --  Each thread gets same seed, a different sequence number, no offset
        hiprand_init(1234, idx, 0, &curandStates[idx]);
    }
}

__device__ char* allocateMemoryPoolGPU()
{
    int blockId = blockIdx.x;
    int startIdx = blockId * PORTIONS_PER_BLOCK;
    for (int i = 0; i < PORTIONS_PER_BLOCK; ++i) {
        if (atomicCAS(&allocationMapGPU[startIdx + i], 0, 1) == 0) {
            // Memory portion allocated
            return &memoryPool[(startIdx + i) * PORTION_SIZE];
        }
    }
    return NULL; // Allocation failed  
}

__device__ void deallocateMemoryPoolGPU(void* vptr)
{
    char *ptr = (char*)vptr;
    int idx = (ptr - memoryPool) / PORTION_SIZE;
    // Mark as free
    atomicExch(&allocationMapGPU[idx], 0);
}
/* -- End -- Function in Version 1.3*/

/* Function in version 1.3.1*/
__device__ char* allocateMemoryPoolGPURandomly()
{
    int blockId = blockIdx.x;
    int startIdx = blockId * PORTIONS_PER_BLOCK;

    while (true) {
        unsigned int randStateIdx = threadIdx.x + blockIdx.x * blockDim.x; // Use thread and block id to access unique hiprandState
        float fi = hiprand_uniform(&curandStates[randStateIdx]);
        int i = (int)(fi * PORTIONS_PER_BLOCK);

        // Generate a random portion ID within this block's range
        int portionId = startIdx + i;

        // Try to atomically set the portion's allocation flag from 0 to 1
        if (atomicCAS(&allocationMapGPU[portionId], 0, 1) == 0) {
            // Memory portion successfully allocated
            return &memoryPool[portionId * PORTION_SIZE];
        }
        // If not successful, the loop continues, trying a new random portion ID
    }

    return NULL; // Allocation failed  
}

__device__ void deallocateMemoryPoolGPURandomly(void* vptr)
{
    char *ptr = (char*)vptr;
    int idx = (ptr - memoryPool) / PORTION_SIZE;
    // Mark as free
    atomicExch(&allocationMapGPU[idx], 0);
}
/* -- End -- Function in version 1.3.1*/

/* Function in version 1.3.2*/
__device__ void initStackUsedPool(StackUsedPool *s)
{
    s->top = -1; // Initialize top to -1 indicating the stack is empty
}

__device__ bool isStackUsedPoolFull(StackUsedPool *s) 
{
    return s->top == MAX_SIZE_STACK_USED_POOL - 1;
}

__device__ bool isStackUsedPoolEmpty(StackUsedPool *s) 
{
    return s->top == -1;
}

__device__ void pushStackUsedPool(StackUsedPool *s, int element)
{
    s->items[++s->top] = element; // Increment top and add the element to the stack
}

__device__ int popStackUsedPool(StackUsedPool *s)
{
    return s->items[s->top--]; // Return the top element and decrement top
}

__device__ char* allocateMemoryPoolGPURandomlyStack(StackUsedPool *s)
{
    int blockId = blockIdx.x;
    int startIdx = blockId * PORTIONS_PER_BLOCK;

    //if the stack is not empty, fetch a index from the stack.
    if(isStackUsedPoolEmpty(s) == 0)
    {
        for(int i = 0; i < MAX_SIZE_STACK_USED_POOL; i++)
        {
            int si = popStackUsedPool(s);
            if (atomicCAS(&allocationMapGPU[si], 0, 1) == 0) 
            {
                // Memory portion successfully allocated
                return &memoryPool[si * PORTION_SIZE];
            }
        }
    }

    while (true) {
        unsigned int randStateIdx = threadIdx.x + blockIdx.x * blockDim.x; // Use thread and block id to access unique hiprandState
        float fi = hiprand_uniform(&curandStates[randStateIdx]);
        int i = (int)(fi * PORTIONS_PER_BLOCK);

        // Generate a random portion ID within this block's range
        int portionId = startIdx + i;

        // Try to atomically set the portion's allocation flag from 0 to 1
        if (atomicCAS(&allocationMapGPU[portionId], 0, 1) == 0) {
            // Memory portion successfully allocated
            return &memoryPool[portionId * PORTION_SIZE];
        }
        // If not successful, the loop continues, trying a new random portion ID
    }

    return NULL; // Allocation failed  
}

__device__ void deallocateMemoryPoolGPURandomlyStack(void* vptr, StackUsedPool *s)
{
    char *ptr = (char*)vptr;
    int idx = (ptr - memoryPool) / PORTION_SIZE;

    if(isStackUsedPoolFull(s) == 0)
    {
        pushStackUsedPool(s, idx);
    }

    // Mark as free
    atomicExch(&allocationMapGPU[idx], 0);
}
/* -- End -- Function in version 1.3.1*/

/*-- Version 1.3.5*/
__device__ int sumAllocatieSize(struct MMUOnTransfer** pMMUOnTransfer, unsigned int sizeAllocate, int *d_sizeTotalallocate, ThreadPointersSizeStore* store)
{
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;

    // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load each thread's data into shared memory
    sdata[tid] = sizeAllocate; // Example: using thread index as data; replace with actual data if needed
    __syncthreads(); // Ensure all threads have written their data to shared memory

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Make sure all additions at one stage are done!
    }

    int index = findPairSeocndPointersNULL(store);
    // Write result for this block to global memory
    if (tid == 0)
    {
        (pMMUOnTransfer[bid])->sizeAllocate = sdata[0];
        (pMMUOnTransfer[bid]) -> syncFlag = 0;
        d_sizeTotalallocate[bid*MAX_SECOND_ALLOCATE + index] = sdata[0];
    } 
    
    return index;
}

//allocation on Each thread
__device__ void* allocateThr(size_t allocateSize, struct MMUOnTransfer** pMMUOnTransfer, int* d_sizetmpAllocate, ThreadPointersSizeStore* store, int *d_sizeTotalallocate)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int total_i = bid * BLOCKSIZE + tid;
    int offset = 0;
    d_sizetmpAllocate[total_i] = allocateSize;
    // pMMUOnTransfer -> sizeAllocate[tid + bid * BLOCKSIZE] = allocateSize;
    // int index = sumAllocatieSize(pMMUOnTransfer, allocateSize, d_sizeTotalallocate, store);

    extern __shared__ int sdata[];

    // Load each thread's data into shared memory
    sdata[tid] = allocateSize; // Example: using thread index as data; replace with actual data if needed
    __syncthreads(); // Ensure all threads have written their data to shared memory

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Make sure all additions at one stage are done!
    }

    int index = findPairSeocndPointersNULL(store);
    // Write result for this block to global memory
    if (tid == 0)
    {
        // printf("iii%d\n", index);
        (pMMUOnTransfer[bid])->sizeAllocate = sdata[0];
        (pMMUOnTransfer[bid]) -> syncFlag = 0;
        d_sizeTotalallocate[bid*MAX_SECOND_ALLOCATE + index] = sdata[0];
    } 
    //store the startOffset
    __syncthreads();
    if(tid == 0)
    {
        //calculate each thread's offset
        int prev = d_sizetmpAllocate[BLOCKSIZE*bid];
        d_sizetmpAllocate[BLOCKSIZE*bid] = 0;

        for (int i = 1; i < BLOCKSIZE; ++i)
        {
            int temp = d_sizetmpAllocate[BLOCKSIZE*bid+i];
            d_sizetmpAllocate[BLOCKSIZE*bid+i] = prev + d_sizetmpAllocate[BLOCKSIZE*bid+i-1];
            prev = temp;
        }

        while (atomicAdd(&((pMMUOnTransfer[bid]) -> syncFlag), 0) != 1) 
        {
        }

        // while((pMMUOnTransfer+bid) -> addressAllocate == NULL)
        // {

        // }
    }

    __syncthreads();

    // Ensure all threads have written their allocating size to transfer structure
    // __syncthreads();
    
    //after all threads finish send the allocating size to transfer structure,
    //thread 0 sets the flag to zero, to notify CPU to calculate the allocation.
    // if(tid == 0)
    // {
    //     // atomicSub(&(pMMUOnTransfer -> syncFlag[bid]), 1);
    //     //allocation tag
    //     pMMUOnTransfer -> syncFlag[bid] = 0;
    // }
    
    // Ensure threads o have set the flag successfully
    // __syncthreads();
    // Wait for the host to finish allocation and set the continue flag

    //read the allocation adderess
    offset = d_sizetmpAllocate[total_i];

    void* tmpStore =  (void*)((char*)(pMMUOnTransfer[bid]) -> addressAllocate + offset);
    
    insertPairSeocndPointersPos(store, tmpStore, allocateSize, index);
    d_sizetmpAllocate[total_i] = 0;
    if(tid == 0)
    {
        (pMMUOnTransfer[bid]) -> addressAllocate = NULL;
    }

    return tmpStore;
}

__device__ void initializeStoreSecond(ThreadPointersSizeStore* store) 
{
    for(int i = 0; i < MAX_SECOND_ALLOCATE; i++)
    {
        store -> pairs[i].key = NULL;
    }
}

__device__ int insertPairSeocndPointers(ThreadPointersSizeStore* store, void* key, int value) {
    // Insert new key-value pair
    for(int i = 0; i < MAX_SECOND_ALLOCATE; i++)
    {
        if(store->pairs[i].key == NULL)
        {
            store->pairs[store->size].key = key;
            store->pairs[store->size].sizeAllocate = value;
            store->size++;
            return i;
        }
    }
}

__device__ int findPairSeocndPointersNULL(ThreadPointersSizeStore* store)
{
    for(int i = 0; i < MAX_SECOND_ALLOCATE; i++)
    {
        if(store->pairs[i].key == NULL)
        {
            return i;
        }
    }
}

__device__ void insertPairSeocndPointersPos(ThreadPointersSizeStore* store, void* key, int value, int index)
{
    store->pairs[index].key = key;
    store->pairs[index].sizeAllocate = value;
    store->size++;
}

__device__ int deletePair(ThreadPointersSizeStore* store, void* key, int* index) {
    for (int i = 0; i < MAX_SECOND_ALLOCATE; i++) {
        if (store->pairs[i].key == key) {
            store->pairs[i].key = NULL;
            int ReValue = store->pairs[i].sizeAllocate;
            // // Found key, now delete it by shifting the rest of the array
            // for (int j = i; j < store->size - 1; j++) {
            //     store->pairs[j] = store->pairs[j + 1];
            // }
            *index = i;
            store->size--;
            return ReValue; // Success
        }
    }
    // printf("Key not found.\n");
    // return -1; // Failure
}

__device__ void freeThr(void* freeAddress, struct MMUOnTransfer** pMMUOnTransfer, ThreadPointersSizeStore* store, int *d_sizeTotalallocate, int* d_sizetmpAllocate)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    //clean the store
    int index = -1;
    int sizeAllocate = deletePair(store, freeAddress, &index);
    // if(tid == 0)
    // {
    //     printf("size--- %d\n",index);
    // }
    // printf("index %d\n", index);

    atomicSub(d_sizeTotalallocate + bid * MAX_SECOND_ALLOCATE + index, sizeAllocate);
    __syncthreads();

    if(tid == 0)
    {
        // printf("size %d\n",d_sizeTotalallocate[bid * MAX_SECOND_ALLOCATE + index]);
        // printf("tid %d %d\n", tid, bid);
        if(d_sizeTotalallocate[bid * MAX_SECOND_ALLOCATE + index] == 0)
        {
            (pMMUOnTransfer[bid]) -> addressFree = freeAddress;
            (pMMUOnTransfer[bid]) -> syncFlag = 2;
            // printf("addFree %d, %d\n",sizeAllocate, index);

            while (atomicAdd(&((pMMUOnTransfer[bid]) -> syncFlag), 0) == 2) 
            {
            }
        }
    }
    // Ensure all threads have written their allocating size to transfer structure
    __syncthreads();
    
    // Wait for the host to finish allocation and set the continue flag
   
}
/*-- End -- Version 1.3.5*/