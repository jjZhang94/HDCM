#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        // 查询最大block size
        printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
        // 如果需要，还可以查询其他相关的限制
        printf("  Max threads dimensions (x, y, z): (%d, %d, %d)\n",
               prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max grid dimensions (x, y, z): (%d, %d, %d)\n",
               prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
    return 0;
}
